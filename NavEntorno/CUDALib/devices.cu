#include "CUDAlib.h"
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

void enumerateDevices() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    printf("Devices: %d\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        printf("%s\n", deviceProp.name);
    }
}