#include "hip/hip_runtime.h"
#include "CUDAlib.h"
#include <cutil_inline.h>

using namespace std;

float * dV1;
float * dV2;
float * dV3;

__global__
void vecAdd(float * v1, float * v2, float * v3, float N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    //if (i < N)
        v3[i] = v1[i] + v2[i];
}

void sumaArrays() {

    int N = 640 * 480;
    size_t size = N * sizeof(float);
    float * v1 = (float *)malloc(size);
    float * v2 = (float *)malloc(size);
    float * v3 = (float *)malloc(size);

    for (int i = 0; i < N; i++) {
        v1[i] = 1;
        v2[i] = 2;
    }

    cutilSafeCall(hipMalloc(&dV1, size));
    cutilSafeCall(hipMalloc(&dV2, size));
    cutilSafeCall(hipMalloc(&dV3, size));

    cutilSafeCall(hipMemcpy(dV1, v1, size, hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(dV2, v2, size, hipMemcpyHostToDevice));

    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vecAdd<<<blocksPerGrid, threadsPerBlock>>> (dV1, dV2, dV3, N);

    cutilSafeCall(hipMemcpy(v1, dV1, size, hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(v2, dV2, size, hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(v3, dV3, size, hipMemcpyDeviceToHost));

    cutilSafeCall(hipFree(dV1));
    cutilSafeCall(hipFree(dV2));
    cutilSafeCall(hipFree(dV3));

    for (int i = 0; i < N; i++) {
        printf("%d: %f\t%f\t%f\n", i, v1[i], v2[i], v3[i]);
    }

    free(v1);
    free(v2);
    free(v3);
}