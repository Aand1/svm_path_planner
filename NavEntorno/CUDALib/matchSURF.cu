#include "hip/hip_runtime.h"
#include "CUDAlib.h"
#include <cutil_inline.h>
#include <c++/4.4/limits>
#include <c++/4.4/cmath>

__constant__ float * d_desc1;
__constant__ float * d_desc2;

__global__
void calcMeanSdv(float * desc1, float * desc2, float * m1, float * m2, float * sdv1, float * sdv2, int n1, int n2) {
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    float * m;
    float * sdv;
    float * desc;

    if (pos < n1) {
        m = m1;
        sdv = sdv1;
        desc = desc1;
    } else if (pos < (n1 + n2)) {
        pos = pos - n1;
        m = m2;
        sdv = sdv2;
        desc = desc2;        
    } else return;
    
    m[pos] = 0.0f;
    sdv[pos] = 0.0f;

    #pragma unroll 64
    for (int i = 0; i < SURF_DESCRIPTOR_SIZE; i++) {
        m[pos] = __fadd_rn(m[pos], desc[pos * SURF_DESCRIPTOR_SIZE + i]);
    }
    m[pos] = __fdiv_rn(m[pos], SURF_DESCRIPTOR_SIZE);


    float tmp;
    for (int i = 0; i < SURF_DESCRIPTOR_SIZE; i++) {
        desc[pos * SURF_DESCRIPTOR_SIZE + i] = __fadd_rn(desc[pos * SURF_DESCRIPTOR_SIZE + i], -m[pos]);
        tmp = __fmul_rn(desc[pos * SURF_DESCRIPTOR_SIZE + i], desc[pos * SURF_DESCRIPTOR_SIZE + i]);

        sdv[pos] = __fadd_rn(tmp, sdv[pos]);
    }
    sdv[pos] = __fdiv_rn(sdv[pos], SURF_DESCRIPTOR_SIZE);
    sdv[pos] = __fsqrt_rn(sdv[pos]);
}

__global__
void calcMean(float * desc1, float * desc2, float * m1, float * m2) {
    __shared__ float partialSum1[MEAN_SDV_THREADS][SURF_DESCRIPTOR_SIZE];
    __shared__ float partialSum2[MEAN_SDV_THREADS][SURF_DESCRIPTOR_SIZE];
    
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;

    unsigned int descPos = (bx * (MEAN_SDV_THREADS) + ty) * SURF_DESCRIPTOR_SIZE + tx;
    unsigned int mPos = bx * MEAN_SDV_THREADS + ty;

    partialSum1[ty][tx] = desc1[descPos];
    partialSum2[ty][tx] = desc2[descPos];

    for (unsigned int stride = blockDim.x>>1; stride > 0; stride >>= 1) {
        __syncthreads();

        if (tx < stride) {
            partialSum1[ty][tx] += partialSum1[ty][tx + stride];
            partialSum2[ty][tx] += partialSum2[ty][tx + stride];
        }        
    }

    __syncthreads();

    if (tx == 0) {
        m1[mPos] = partialSum1[ty][0] / SURF_DESCRIPTOR_SIZE;
        m2[mPos] = partialSum2[ty][0] / SURF_DESCRIPTOR_SIZE;
    }    
}

__global__
void calcSdv(float * desc1, float * desc2, float * m1, float * m2, float * sdv1, float * sdv2) {
    __shared__ float partialSum1[MEAN_SDV_THREADS][SURF_DESCRIPTOR_SIZE];
    __shared__ float partialSum2[MEAN_SDV_THREADS][SURF_DESCRIPTOR_SIZE];
    __shared__ float mean1[MEAN_SDV_THREADS];
    __shared__ float mean2[MEAN_SDV_THREADS];

    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;

    unsigned int mPos = bx * MEAN_SDV_THREADS + ty;
    unsigned int descPos = (bx * (MEAN_SDV_THREADS) + ty) * SURF_DESCRIPTOR_SIZE + tx;

    if (tx == 0) {
        mean1[ty] = m1[mPos];
        mean2[ty] = m2[mPos];
    }
    __syncthreads();

    float sub1 = desc1[descPos] - mean1[ty];
    float sub2 = desc2[descPos] - mean2[ty];

    partialSum1[ty][tx] = sub1 * sub1;
    partialSum2[ty][tx] = sub2 * sub2;
    
    for (unsigned int stride = blockDim.x>>1; stride > 0; stride >>= 1) {
        __syncthreads();

        if (tx < stride) {
            partialSum1[ty][tx] += partialSum1[ty][tx + stride];
            partialSum2[ty][tx] += partialSum2[ty][tx + stride];
        }
    }

    __syncthreads();

    if (tx == 0) {
        sdv1[mPos] = sqrt(partialSum1[ty][0] / SURF_DESCRIPTOR_SIZE);
        sdv2[mPos] = sqrt(partialSum2[ty][0] / SURF_DESCRIPTOR_SIZE);
    }       
}

__global__
void calcCorrelation(float * desc1, float * desc2, float * corr, float * mean1, float * mean2, float * sdv1, float * sdv2, bool * resp1, bool * resp2, int rows, int cols) {

    __shared__ float tmpDesc1[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tmpDesc2[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tmpMean[2][TILE_WIDTH];
    __shared__ float tmpSdv[2][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    int width = SURF_DESCRIPTOR_SIZE;

    if (ty == 0) {
        tmpMean[ty][tx] = mean1[by * TILE_WIDTH + tx];
        tmpSdv[ty][tx] = sdv1[by * TILE_WIDTH + tx];
    } else if (ty == 1) {
        tmpMean[ty][tx] = mean2[bx * TILE_WIDTH + tx];
        tmpSdv[ty][tx] = sdv2[bx * TILE_WIDTH + tx];
    }
    __syncthreads();

    float pVal = 0;
    for (int m = 0; m < width / TILE_WIDTH; m++) {
        // Collaborative loading of tiles
        tmpDesc1[ty][tx] = desc1[row * width + (m * TILE_WIDTH + tx)];
        tmpDesc2[tx][ty] = desc2[col * width + (m * TILE_WIDTH + ty)];

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            pVal += (tmpDesc1[ty][k] - tmpMean[0][ty]) * (tmpDesc2[tx][k] - tmpMean[1][tx]);
        }

        __syncthreads();
    }
    pVal /= (width - 1) * tmpSdv[0][ty] * tmpSdv[1][tx];
        
    corr[row * cols + col] = pVal;
}

__global__
void bestCorrX(float * corr, int * bestCorr1, int size1, int size2) {
    __shared__ float partialComp[BEST_CORR_X][BEST_CORR_Y];
    __shared__ int partialIdx[BEST_CORR_X][BEST_CORR_Y];

    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    float tmpVal = 0;

    unsigned int partialCorrPos = (bx * BEST_CORR_Y) + tx;
    unsigned int yPos = 0;

    partialComp[ty][tx] = corr[(ty * size2) + partialCorrPos];
    partialIdx[ty][tx] = ty;

    // Obtenemos el mayor de cada "pseudobloque"
    for (unsigned int i = BEST_CORR_X; i < size1; i += BEST_CORR_X) {
        yPos = i + ty;
        tmpVal = corr[(yPos * size2) + partialCorrPos];
        if (partialComp[ty][tx] < tmpVal) {
            partialComp[ty][tx] = tmpVal;
            partialIdx[ty][tx] = yPos;
        }
    }

    // Obtenemos el mayor para el bloque inicial
    for (unsigned int stride = blockDim.y>>1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (ty < stride) {
            if (partialComp[ty + stride][tx] > partialComp[ty][tx]) {
                partialComp[ty][tx] = partialComp[ty + stride][tx];
                partialIdx[ty][tx] = partialIdx[ty + stride][tx];
            }
        }        
    }

    if (ty == 0) {
        bestCorr1[partialCorrPos] = partialIdx[ty][tx];
    }
}

__global__
void calcBestCorr(float * corr, int * bestCorr1, int * bestCorr2, int rows, int cols) {
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    int maxPos;
    
    float * bestCorrelation;

    if (pos < rows) {
        int row = pos;
        int best = -1;
        float bestCorr = 0.0f;
        //#pragma unroll 30
        for (int i = row * cols; i < (row * cols) + cols; i++) {
            //if ((corr[i] > bestCorr) && (corr[i] > CORRELATION_THRESH)) {
            if (corr[i] > bestCorr) {
                best = i - (row * cols);
                bestCorr = corr[i];
            }
        }
        bestCorr1[row] = best;
    } else if (pos < (rows + cols)) {
        int col = pos - rows;

        int best = 0;
        float bestCorr = 0.0f;
        //#pragma unroll 30
        int tmpPos = 0;
        for (int i = col; i < col + (rows * cols); i += cols) {
            tmpPos++;
            //if ((corr[i] > bestCorr) && (corr[i] > CORRELATION_THRESH)) {
            if (corr[i] > bestCorr) {
                best = (i - col) / cols;
                bestCorr = corr[i];
            }
        }
        bestCorr2[col] = best;
    }
}

__global__
void calcMatches(int * bestCorr1, int * bestCorr2, int * matches, int n1) {
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos >= n1) return;
    
    if (bestCorr2[bestCorr1[pos]] == pos) {
        matches[pos] = bestCorr1[pos];
    } else {
        matches[pos] = -1;
    }
}

void calcMeanSdvSequential(vector<t_Point> points1, vector<t_Point> points2, vector<float> desc1, vector<float> desc2, float * d_m1, float * d_m2, float * d_sdv1, float * d_sdv2) {
    float* avg1 = (float*) malloc(sizeof (float) * points1.size());
    float* avg2 = (float*) malloc(sizeof (float) * points2.size());
    float* dev1 = (float*) malloc(sizeof (float) * points1.size());
    float* dev2 = (float*) malloc(sizeof (float) * points2.size());

    int descriptor_size = 64;
    for (int i = 0; i < points1.size(); i++) {
        // find average and standard deviation of each descriptor
        avg1[i] = 0;
        dev1[i] = 0;
        
        for (int k = i * descriptor_size; k < (i * descriptor_size) + descriptor_size; k++) {
            avg1[i] += desc1.at(k);
        }        
        avg1[i] /= descriptor_size;
        for (int k = i * descriptor_size; k < (i * descriptor_size) + descriptor_size; k++) {
            dev1[i] += (desc1.at(k) - avg1[i]) * (desc1.at(k) - avg1[i]);
        }
        dev1[i] = sqrt(dev1[i] / descriptor_size);
    }
    for (int i = 0; i < points2.size(); i++) {
        // find average and standard deviation of each descriptor
        avg2[i] = 0;
        dev2[i] = 0;

        for (int k = i * descriptor_size; k < (i * descriptor_size) + descriptor_size; k++) {
            avg2[i] += desc2.at(k);
        }
        avg2[i] /= descriptor_size;
        for (int k = i * descriptor_size; k < (i * descriptor_size) + descriptor_size; k++) {
            dev2[i] += (desc2.at(k) - avg2[i]) * (desc2.at(k) - avg2[i]);
        }
        dev2[i] = sqrt(dev2[i] / descriptor_size);
    }

    for (int i = 0; i < 65; i++) {
        cout << i << "[" << dev2[i] << "]";
    }
    cout << endl;

    cutilSafeCall(hipMemcpy(d_m1, avg1, points1.size() * sizeof(float), hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_m2, avg2, points2.size() * sizeof(float), hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_sdv1, dev1, points1.size() * sizeof(float), hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_sdv2, dev2, points2.size() * sizeof(float), hipMemcpyHostToDevice));

    delete avg1;
    delete avg2;
    delete dev1;
    delete dev2;
}

template<typename T>
inline bool isNAN(T value) {
    return std::numeric_limits<T>::has_quiet_NaN;
}

void bruteMatchParallel(vector<t_Point> points1, vector<t_Point> points2, vector<float> desc1, vector<float> desc2, vector<int> &matches, t_Timings &timings) {

    cout << "Points1 " << points1.size() << endl;
    cout << "Points2 " << points2.size() << endl;

    clock_t myTime = clock();    
    int size1 = (int(points1.size() / TILE_WIDTH) + 1) * TILE_WIDTH;
    int size2 = (int(points2.size() / TILE_WIDTH) + 1) * TILE_WIDTH;
    int size = max(size1, size2);
    size_t corrSize = size1 * size2 * sizeof(float);
    cout << "corrSize = " << (size1 * size2) << endl;

    float * h_desc1 = (float *)malloc(size * SURF_DESCRIPTOR_SIZE * sizeof(float));
    float * h_desc2 = (float *)malloc(size * SURF_DESCRIPTOR_SIZE * sizeof(float));
    float * h_response1 = (float *)malloc(size1 * sizeof(float));
    float * h_response2 = (float *)malloc(size2 * sizeof(float));    

    for (int i = 0; i < desc1.size(); i++) {        
        h_desc1[i] = (float)desc1.at(i);
    }
    for (int i = 0; i < desc2.size(); i++) {
        h_desc2[i] = (float)desc2.at(i);
    }
    for (int i = desc1.size(); i < size * SURF_DESCRIPTOR_SIZE; i++) {
        h_desc1[i] = 0;
    }
    for (int i = desc2.size(); i < size * SURF_DESCRIPTOR_SIZE; i++) {
        h_desc2[i] = 0;
    }
    for (int i = 0; i < points1.size(); i++) {
        h_response1[i] = points1.at(i).response;
    }
    for (int i = 0; i < points2.size(); i++) {
        h_response2[i] = points2.at(i).response;
    }

    float * d_corr;
    float * d_m1;
    float * d_m2;
    float * d_sdv1;
    float * d_sdv2;
    bool * d_response1;
    bool * d_response2;
    int * d_bestCorr1;
    int * d_bestCorr2;
    int * d_matches;        

    //cutilSafeCall(hipMalloc(&d_m2, size1 + size2 * sizeof(float)));
    //cutilSafeCall(hipMalloc(&d_sdv2, size2 * sizeof(float)));
    cutilSafeCall(hipMalloc(&d_m1, size * sizeof(float)));
    cutilSafeCall(hipMalloc(&d_sdv1, size * sizeof(float)));
    cutilSafeCall(hipMalloc(&d_m2, size * sizeof(float)));
    cutilSafeCall(hipMalloc(&d_sdv2, size * sizeof(float)));
    cutilSafeCall(hipMalloc(&d_desc1, size * SURF_DESCRIPTOR_SIZE * sizeof(float)));
    cutilSafeCall(hipMalloc(&d_desc2, size * SURF_DESCRIPTOR_SIZE * sizeof(float)));

    cutilSafeCall(hipMemcpy(d_desc1, h_desc1, size * SURF_DESCRIPTOR_SIZE * sizeof(float), hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_desc2, h_desc2, size * SURF_DESCRIPTOR_SIZE * sizeof(float), hipMemcpyHostToDevice));

    timings.tMalloc1 = clock() - myTime;
    myTime = clock();

    int threadsPerBlock = 512;
    int blocksPerGrid = ((points1.size() + points2.size()) / threadsPerBlock) + 1;
    //calcMeanSdv <<< blocksPerGrid, threadsPerBlock >>> (d_desc1, d_desc2, d_m1, d_m2, d_sdv1, d_sdv2, points1.size(), points2.size());
    //hipDeviceSynchronize();
    //calcMeanSdvSequential(points1, points2, desc1, desc2, d_m1, d_m2, d_sdv1, d_sdv2);
    dim3 dimBlockMeanSdv(SURF_DESCRIPTOR_SIZE, MEAN_SDV_THREADS);
    dim3 dimGridMeanSdv(size / dimBlockMeanSdv.y, 1);    
    calcMean <<< dimGridMeanSdv, dimBlockMeanSdv >>> (d_desc1, d_desc2, d_m1, d_m2);
    calcSdv <<< dimGridMeanSdv, dimBlockMeanSdv >>> (d_desc1, d_desc2, d_m1, d_m2, d_sdv1, d_sdv2);
    hipDeviceSynchronize();

    /*float * m1 = (float *)malloc(size * sizeof(float));
    cout << dimBlockMeanSdv.x << endl;
    
    cutilSafeCall(hipMemcpy(m1, d_sdv2, size * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < 65; i++) {
        cout << i << "[" << m1[i] << "]";
    }
    cout << endl;
    free(m1);//*/

    timings.tCalcMeanSdv = clock() - myTime;
    myTime = clock();

    //cutilSafeCall(hipFree(d_m1));
    //cutilSafeCall(hipFree(d_m2));

    cutilSafeCall(hipMalloc(&d_corr, corrSize));
    cutilSafeCall(hipMalloc(&d_response1, size1 * sizeof(bool)));
    cutilSafeCall(hipMalloc(&d_response2, size2 * sizeof(bool)));
    cutilSafeCall(hipMalloc(&d_bestCorr1, size1 * sizeof(int)));
    cutilSafeCall(hipMalloc(&d_bestCorr2, size2 * sizeof(int)));
    cutilSafeCall(hipMalloc(&d_matches, size1 * sizeof(int)));

    cutilSafeCall(hipMemcpy(d_response1, h_response1, size1 * sizeof(bool), hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_response2, h_response2, size2 * sizeof(bool), hipMemcpyHostToDevice));

    timings.tMalloc2 = clock() - myTime;
    timings.tMalloc = timings.tMalloc1 + timings.tMalloc2;
    myTime = clock();

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(size2 / TILE_WIDTH, size1 / TILE_WIDTH);

    calcCorrelation <<< dimGrid, dimBlock >>> (d_desc1, d_desc2, d_corr, d_m1, d_m2, d_sdv1, d_sdv2, d_response1, d_response2, size1, size2);
    hipDeviceSynchronize();

    /*float * hCorr = (float *)malloc(corrSize);
    cutilSafeCall(hipMemcpy(hCorr, d_corr, corrSize, hipMemcpyDeviceToHost));

    for (int i = 0; i < 16; i++) {
        cout << i << " ";
        for (int j = 0; j < 32; j++) {
            cout << j << "[" << hCorr[i * size2 + j] << "]";
        }
        cout << endl;
    }
    cout << endl;
    //free(hCorr);//*/

    timings.tCalcCorrelation = clock() - myTime;
    myTime = clock();

    //calcBestCorr <<< blocksPerGrid, threadsPerBlock >>> (d_corr, d_bestCorr1, d_bestCorr2, points1.size(), points2.size());
    dim3 dimBlockBestCorrX(BEST_CORR_Y, BEST_CORR_X);
    dim3 dimGridBestCorrX(size1 / BEST_CORR_Y, 1);
    bestCorrX <<< dimGridBestCorrX, dimBlockBestCorrX >>> (d_corr, d_bestCorr2, size1, size2);
    hipDeviceSynchronize();

    timings.tCalcBestCorr = clock() - myTime;
    myTime = clock();

    int * bestCorr2 = (int *)malloc(size1 * sizeof(int));
    cutilSafeCall(hipMemcpy(bestCorr2, d_bestCorr2, size1 * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < points2.size(); i++) {
        cout << "[" << bestCorr2[i] << "]";
    }
    cout << endl;    
    free(bestCorr2);//*/

    /*blocksPerGrid = ((points1.size() - 1) / threadsPerBlock) + 1;
    calcMatches <<< blocksPerGrid, threadsPerBlock >>> (d_bestCorr1, d_bestCorr2, d_matches, points1.size());
    hipDeviceSynchronize();

    timings.tCalcMatches = clock() - myTime;
    myTime = clock();

    int * h_matches = (int *)malloc(points1.size() * sizeof(int));
    cutilSafeCall(hipMemcpy(h_matches, d_matches, points1.size() * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < points1.size(); i++)
        matches.push_back(h_matches[i]);

    timings.tMemCpy = clock() - myTime;*/
    myTime = clock();
    
    cutilSafeCall(hipFree(d_m1));
    cutilSafeCall(hipFree(d_m2));
    cutilSafeCall(hipFree(d_desc1));
    cutilSafeCall(hipFree(d_desc2));
    cutilSafeCall(hipFree(d_corr));
    cutilSafeCall(hipFree(d_sdv1));
    cutilSafeCall(hipFree(d_sdv2));
    cutilSafeCall(hipFree(d_response1));
    cutilSafeCall(hipFree(d_response2));
    cutilSafeCall(hipFree(d_bestCorr1));
    cutilSafeCall(hipFree(d_bestCorr2));
    cutilSafeCall(hipFree(d_matches));

    timings.tFreeMem = clock() - myTime;
    
    free(h_desc1);
    free(h_desc2);
    free(h_response1);
    free(h_response2);
    //free(h_matches);

    
}
