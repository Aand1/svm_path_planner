/*
    Copyright 2013 Néstor Morales Hernández <nestor@isaatc.ull.es>

    Licensed under the Apache License, Version 2.0 (the "License");
    you may not use this file except in compliance with the License.
    You may obtain a copy of the License at

        http://www.apache.org/licenses/LICENSE-2.0

    Unless required by applicable law or agreed to in writing, software
    distributed under the License is distributed on an "AS IS" BASIS,
    WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
    See the License for the specific language governing permissions and
    limitations under the License.
*/

#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <stdio.h>
#include <iostream>
#include <complex>

#define BLOCK_SIZE 1024
#define MEM_BLOCK 1024

using namespace std;

#  define CUDA_SAFE_CALL_NO_SYNC( call) do {                                \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } } while (0)

#  define CUDA_SAFE_CALL( call) do {                                        \
    CUDA_SAFE_CALL_NO_SYNC(call);                                            \
    hipError_t err = hipDeviceSynchronize();                                 \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } } while (0)

#  define CUDA_SAFE_CALL_NO_SYNC( call) call
#  define CUDA_SAFE_CALL( call) call

__global__
void checkEdge(const float2 * d_pointsInMap, const unsigned int nPointsInMap,
               const float2 * d_edgeU, const float2 * d_edgeV, const unsigned int nEdges,
               float minDist, bool * d_validEdges) {
               
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx > nEdges)
        return;
    
    __syncthreads();

    float2 v = d_edgeU[idx];
    float2 w = d_edgeV[idx];
    float2 p;
    
    float lineLenghtSqr;
    float t;
    float tmpDist;
    
    float2 tmpPoint;
    bool valid = true;
    for (unsigned int i = 0; i < nPointsInMap; i++) {
        p = d_pointsInMap[i];
        
        lineLenghtSqr = (v.x - w.x) * (v.x - w.x) + (v.y - w.y) * (v.y - w.y);
        
        if (lineLenghtSqr == 0) {
            tmpDist = (p.x - v.x) * (p.x - v.x) + (p.y - v.y) * (p.y - v.y);
            if (tmpDist < minDist) {
                valid = false;
                break;
            }
            continue;
        }
        
        t = ((p.x - v.x) * (w.x - v.x) + (p.y - v.y) * (w.y - v.y)) / lineLenghtSqr;

        if (t < 0) {
            tmpDist = (p.x - v.x) * (p.x - v.x) + (p.y - v.y) * (p.y - v.y);
            if (tmpDist < minDist) {
                valid = false;
                break;
            }
            continue;
        }
        if (t > 1) {
            tmpDist = (p.x - w.x) * (p.x - w.x) + (p.y - w.y) * (p.y - w.y);
            if (tmpDist < minDist) {
                valid = false;
                break;
            }
            continue;
        }
        
        tmpPoint = make_float2(v.x + t * (w.x - v.x), v.y + t * (w.y - v.y));
    
        tmpDist = (p.x - tmpPoint.x) * (p.x - tmpPoint.x) + (p.y - tmpPoint.y) * (p.y - tmpPoint.y);
        
        
        if (tmpDist < minDist) {
            valid = false;
            break;
        }
    }
    
    __syncthreads();
    
    d_validEdges[idx] = valid;
}

extern "C"
void launchCheckEdges(const float2 * &h_pointsInMap, const unsigned int &nPointsInMap,
                      const float2 * &h_edgeU, const float2 * &h_edgeV, const unsigned int &nEdges,
                      const float &minDist, bool * &h_validEdges) {
                      
    float2 *d_pointsInMap, *d_edgeU, *d_edgeV;
    bool *d_validEdges;
    CUDA_SAFE_CALL(hipMalloc(&d_pointsInMap, (sizeof(float2) * nPointsInMap)));
    CUDA_SAFE_CALL(hipMalloc(&d_edgeU, (sizeof(float2) * nEdges)));
    CUDA_SAFE_CALL(hipMalloc(&d_edgeV, (int)(sizeof(float2) * nEdges)));
    CUDA_SAFE_CALL(hipMalloc(&d_validEdges, (int)(sizeof(bool) * nEdges)));
    
    CUDA_SAFE_CALL(hipMemcpy(d_pointsInMap, h_pointsInMap, sizeof(float2) * nPointsInMap, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_edgeU, h_edgeU, sizeof(float2) * nEdges, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_edgeV, h_edgeV, sizeof(float2) * nEdges, hipMemcpyHostToDevice));
    
    const dim3 blockSize(BLOCK_SIZE, 1, 1);
    const dim3 gridSize((nEdges / blockSize.x) + 1, 1, 1);
    
    struct timespec start, finish;
    double elapsed;
    
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    checkEdge <<<gridSize, blockSize>>> (d_pointsInMap, nPointsInMap, d_edgeU, d_edgeV, nEdges, minDist, d_validEdges);
    hipDeviceSynchronize(); CUDA_SAFE_CALL(hipGetLastError());
    
    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    
    std::cout << "Elapsed time for cleaning kernel = " << elapsed << endl;
    
    CUDA_SAFE_CALL(hipMemcpy(h_validEdges, d_validEdges, sizeof(bool) * nEdges, hipMemcpyDeviceToHost));
    
    CUDA_SAFE_CALL(hipFree(d_pointsInMap));
    CUDA_SAFE_CALL(hipFree(d_edgeU));
    CUDA_SAFE_CALL(hipFree(d_edgeV));
    CUDA_SAFE_CALL(hipFree(d_validEdges));
    
    CUDA_SAFE_CALL(hipGetLastError());

}